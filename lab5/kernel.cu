#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <vector>
#include <functional>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <chrono>
#include <random>

using namespace std;

__device__ const double G = 6.67E-11;

ostream &operator<<(ostream &stream, const vector<double> &vec)
{
    for (auto it = vec.begin(); it != --vec.end(); ++it)
    {
        stream << setw(20) << *it;
    }
    stream << setw(20) << vec.back();

    return stream;
}

vector<double> operator*(const vector<double> &lhs, const double &rhs)
{
    vector<double> result(lhs.size());
    for (int i = 0; i < lhs.size(); ++i)
        result[i] = lhs[i] * rhs;
    return result;
}

vector<double> operator*(const double &lhs, const vector<double> &rhs)
{
    return rhs * lhs;
}

vector<double> operator+(const vector<double> &lhs, const vector<double> &rhs)
{
    vector<double> result(lhs.size());
    for (int i = 0; i < lhs.size(); ++i)
        result[i] = lhs[i] + rhs[i];
    return result;
}

vector<double> &operator+=(vector<double> &lhs, const vector<double> &rhs)
{
    for (int i = 0; i < lhs.size(); ++i)
        lhs[i] += rhs[i];
    return lhs;
}

__host__ __device__ double distance3(double x1, double y1, double z1, double x2, double y2, double z2)
{
    return pow(pow(x2 - x1, 2) + pow(y2 - y1, 2) + pow(z2 - z1, 2), 3.0 / 2);
}

vector<double> func(vector<double> &posvel, const vector<double> &masses)
{
    double eps = 1E-15;
    size_t n = masses.size();
    vector<double> result(6 * n);
    for (int i = 0; i < n; ++i)
    {
        result[3 * i] = posvel[3 * i + 3 * n];
        result[3 * i + 1] = posvel[3 * i + 3 * n + 1];
        result[3 * i + 2] = posvel[3 * i + 3 * n + 2];
        double ax(0), ay(0), az(0);
        for (int j = 0; j < n; ++j)
        {
            double coef = -G * masses[j] / max(distance3(posvel[3 * i], posvel[3 * i + 1], posvel[3 * i + 2], posvel[3 * j], posvel[3 * j + 1], posvel[3 * j + 2]), eps);
            ax += (posvel[3 * i] - posvel[3 * j]) * coef;
            ay += (posvel[3 * i + 1] - posvel[3 * j + 1]) * coef;
            az += (posvel[3 * i + 2] - posvel[3 * j + 2]) * coef;
        }
        result[3 * i + 3 * n] = ax;
        result[3 * i + 3 * n + 1] = ay;
        result[3 * i + 3 * n + 2] = az;
    }
    return result;
}

__global__ void funcpar(double *result, double *posvel, double *masses, int n)
{
    double eps = 1E-15;
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n)
    {
        result[3 * idx] = posvel[3 * idx + 3 * n];
        result[3 * idx + 1] = posvel[3 * idx + 3 * n + 1];
        result[3 * idx + 2] = posvel[3 * idx + 3 * n + 2];
        double ax = 0, ay = 0, az = 0;
        for (int j = 0; j < n; ++j)
        {
            double coef = -G * masses[j] / max(distance3(posvel[3 * idx], posvel[3 * idx + 1], posvel[3 * idx + 2], posvel[3 * j], posvel[3 * j + 1], posvel[3 * j + 2]), eps);
            ax += (posvel[3 * idx] - posvel[3 * j]) * coef;
            ay += (posvel[3 * idx + 1] - posvel[3 * j + 1]) * coef;
            az += (posvel[3 * idx + 2] - posvel[3 * j + 2]) * coef;
        }
        result[3 * idx + 3 * n] = ax;
        result[3 * idx + 3 * n + 1] = ay;
        result[3 * idx + 3 * n + 2] = az;
    }
}

__global__ void plustimesequal(double *result, double *rhs, double tau, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n)
    {
        result[idx] += tau * rhs[idx];
    }
}

__global__ void plustimes(double *result, double *lhs, double *rhs, double tau, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n)
    {
        result[idx] = lhs[idx] + tau * rhs[idx];
    }
}

void rungekutta2(const vector<double> &init, const vector<double> &masses, double tmax, double tau, int k, const string &filename)
{
    ofstream output(filename);
    double tcurrent = 0;
    vector<double> pcurrent(init);
    vector<double> k1(init), k2(init);
    output << setprecision(10) << tcurrent << " " << pcurrent << '\n';
    int counter = 0;
    while (tcurrent < tmax)
    {
        k1 = func(pcurrent, masses);
        auto temp = pcurrent + tau / 2 * k1;
        k2 = func(temp, masses);
        pcurrent += tau * k2;
        tcurrent += tau;
        if (++counter % k == 0)
            output << setprecision(10) << tcurrent << " " << pcurrent << '\n';
    }
}

void rungekutta2par(const vector<double> &init, const vector<double> &masses, double tmax, double tau, int k, const string &filename)
{
    ofstream output(filename);
    double tcurrent = 0;
    vector<double> pcurrent(init);
    vector<double> k1(init), k2(init);
    output << setprecision(10) << tcurrent << " " << pcurrent << '\n';

    int counter = 0;
    int size = init.size();
    int msize = masses.size();
    double *pcurrent_dev;
    double *k1_dev;
    double *k2_dev;
    double *masses_dev;
    double *temp_dev;

    hipMalloc(&pcurrent_dev, size * sizeof(double));
    hipMalloc(&k1_dev, size * sizeof(double));
    hipMalloc(&k2_dev, size * sizeof(double));
    hipMalloc(&masses_dev, msize * sizeof(double));
    hipMalloc(&temp_dev, size * sizeof(double));

    hipMemcpy(masses_dev, masses.data(), msize * sizeof(double), hipMemcpyHostToDevice);

    const int bs = 32;

    while (tcurrent < tmax)
    {
        dim3 blocks((msize + bs) / bs, 1, 1);
        dim3 threads(bs, 1, 1);

        hipMemcpy(pcurrent_dev, pcurrent.data(), size * sizeof(double), hipMemcpyHostToDevice);
        funcpar<<<blocks, threads>>>(k1_dev, pcurrent_dev, masses_dev, msize);// calculate k1

        hipMemcpy(k1.data(), k1_dev, size * sizeof(double), hipMemcpyDeviceToHost);
        plustimes<<<(size + bs) / bs, bs>>>(temp_dev, pcurrent_dev, k1_dev, tau / 2, size);// step with k1 on tau/2

        funcpar<<<blocks, threads>>>(k2_dev, temp_dev, masses_dev, msize);//calculate k2
        hipMemcpy(k2.data(), k2_dev, size * sizeof(double), hipMemcpyDeviceToHost);

        hipMemcpy(pcurrent_dev, pcurrent.data(), size * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(k2_dev, k2.data(), size * sizeof(double), hipMemcpyHostToDevice);
        plustimesequal<<<(size + bs) / bs, bs>>>(pcurrent_dev, k2_dev, tau, size);//finally, steps

        hipMemcpy(pcurrent.data(), pcurrent_dev, size * sizeof(double), hipMemcpyDeviceToHost);

        tcurrent += tau;
        if (++counter % k == 0)
            output << setprecision(10) << tcurrent << " " << pcurrent << '\n';
    }
    
    hipFree(pcurrent_dev);
    hipFree(k1_dev);
    hipFree(k2_dev);
    hipFree(masses_dev);
    hipFree(temp_dev);
}

int main(int argc, char **argv)
{
    vector<double> init;
    vector<double> masses;
    
    size_t N = 5000;
    bool seq = False;
 
    init.resize(6 * N);
    masses.resize(N);

    std::random_device seeder;
    const auto seed = seeder.entropy() ? seeder() : time(nullptr);
    std::mt19937 eng(static_cast<std::mt19937::result_type>(seed));
    std::uniform_real_distribution<double> distr1(-10, 10);
    std::uniform_real_distribution<double> distr2(-1, 1);
    std::uniform_real_distribution<double> distr3(0.5, 2.0);
    auto rndpos = std::bind(distr1, eng);
    auto rndvel = std::bind(distr2, eng);
    auto rndmas = std::bind(distr2, eng);

    for (int i = 0; i < 3 * N; ++i)
        init[i] = rndpos();
    for (int i = 3 * N; i < 6 * N; ++i)
        init[i] = rndvel();
    for (int i = 0; i < N; ++i)
        masses[i] = rndmas();

    auto t1 = chrono::high_resolution_clock::now();
    if(seq)
        rungekutta2(init, masses, 0.1, 0.05, 1, "out2");
    auto t2 = chrono::high_resolution_clock::now();

    auto seqtime = chrono::duration_cast<chrono::milliseconds>(t2 - t1).count();
    cout << "Seq time " << seqtime << " ms\n";

    t1 = chrono::high_resolution_clock::now();
    rungekutta2par(init, masses, 0.1, 0.05, 1, "out3");
    t2 = chrono::high_resolution_clock::now();

    auto partime = chrono::duration_cast<chrono::milliseconds>(t2 - t1).count();
    cout << "Cuda time(?) " << partime << " ms\n";

    cout << "acceleration: " << seqtime * 1.0 / partime << "\n";

    return 0;
}
